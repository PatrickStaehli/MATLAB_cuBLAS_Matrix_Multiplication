/*
* cuBLAS_Matrix_Multiplication.cu - MATLAB external interfaces GPU implementation of a matrix multiplication using cuBLAS library.
*
*
* Input:
*	A	- MxN Matrix in float-precision
*	B 	- NxP Matrix in float-precision
* Output:
*	C	- MxP Matrix in float-precision
*
*
*
* The calling syntax from Matlab is:
*
*	C = cuBLAS_Matrix_Multiplication(A,B);
*
* This is a MEX file for MATLAB
*
*
*/


#include "mex.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

/* Input error handling */
/*--------------------------------------------------------------------------------------------------*/
void inputErrorHandling(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
    /* Verify if the inputs have a valid shape and that all requred inputs are given.
     *
     * Expected input from matlab: (MxN real float, NxP real float)
     *
     * Expected output to matlab: MxP real float
     *
    */
  
	// Check if two inputs are given
    if(nrhs != 2)
      mexErrMsgIdAndTxt( "parallel:gpu:cuBLAS_Matrix_Multiplication:invalidNumInputs",
              "Two inputs required.");
			  
    // Check if not more than one output is requested
	if(nlhs > 1)
      mexErrMsgIdAndTxt( "parallel:gpu:cuBLAS_Matrix_Multiplication:maxlhs",
              "Only one output argument of size MxP allowed");
	
	// Check if the input float Array
	if (!mxIsSingle(prhs[0]) || !mxIsSingle(prhs[1]) ) {
        mexErrMsgIdAndTxt("parallel:gpu:cuBLAS_Matrix_Multiplication:InvalidInput", "Input has to be float");
    }
    
    // Check if number of columns of the first input is equal to the number of rows of the second input
	if (mxGetN(prhs[0]) != mxGetM(prhs[1])) {
        mexErrMsgIdAndTxt("parallel:gpu:cuBLAS_Matrix_Multiplication:InvalidInputSize", "column(A) != row(B)");
    }
	
}


/* Host code */
/*--------------------------------------------------------------------------------------------------*/
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){


	//	Variable declarations
	float *device_A, *device_B, *device_C;
	const float *A, *B;
	float *C;
	int num_A_rows, num_A_cols;
	int num_B_rows, num_B_cols;
	int num_C_rows, num_C_cols;


	// Input Error Handling
	inputErrorHandling(nlhs, plhs, nrhs, prhs);
	 
	
	// Read the inputs from Matlab
	A = (float *)mxGetData(prhs[0]);
	B = (float *)mxGetData(prhs[1]);


	// Get the dimension of the input array
	num_A_rows = (int)mxGetM(prhs[0]);
	num_A_cols = (int)mxGetN(prhs[0]);
	num_B_rows = (int)mxGetM(prhs[1]);
	num_B_cols = (int)mxGetN(prhs[1]);
	num_C_rows = num_A_rows;
	num_C_cols = num_B_cols;


	
	// Initialize the output to MATLAB
	plhs[0] = mxCreateNumericMatrix(num_C_rows, num_C_cols, mxSINGLE_CLASS, mxREAL);
	C = (float *)mxGetData(plhs[0]);
    

	// Allocate GPU memory
	hipMalloc(&device_A, sizeof(float) * num_A_rows * num_A_cols);
	hipMalloc(&device_B, sizeof(float) * num_B_rows * num_B_cols);
	hipMalloc(&device_C, sizeof(float) * num_C_rows * num_C_cols);
   
	
	// Create handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	// Set the Matrices
	hipblasSetMatrix(num_A_rows, num_A_cols, sizeof(float), A, num_A_rows, device_A, num_A_rows);
	hipblasSetMatrix(num_B_rows, num_B_cols, sizeof(float), B, num_B_rows, device_B, num_B_rows);

	
	// Scaling facors for matrix Multiplication: C = (alpha*A)*b + (beta*c)
	float alpha = 1.0;
	float beta = 0.0;
	
	
	// Matrix multiplication using cuBLAS: (m X n) * (n X p) = (m X p)
	// Signature: handle, operation, operation, m, n, p, alpha, A, lda, B, ldb, beta, C, ldc
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_A_rows, num_B_cols, num_A_cols, &alpha, device_A, num_A_rows, device_B, num_B_rows, &beta, device_C, num_C_rows);
   
	// Copy C back to host
	hipMemcpy(C,device_C, sizeof(float) * num_C_rows * num_C_cols, hipMemcpyDeviceToHost);        
    

	// Clearing GPU memory cache

    hipblasDestroy(handle);
	hipFree(device_A);
	hipFree(device_B);
	hipFree(device_C);

}
